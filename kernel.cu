#include "cuda_utility.cuh"




int main()
{

	int number_of_steps = 1000;// (int)argv[1];
	std::vector<float> h1(h * w);
	/*	{
		0,0,0,0,0,0,
		0,0,1,0,0,0,
		1,0,1,0,0,0,
		0,1,1,0,0,0,
		0,0,0,0,0,0,
		0,0,0,0,0,0

	};*/


	std::random_device rd{};
	std::mt19937 mersenne_engine{ rd() };  // Generates random integers
	std::uniform_real_distribution<float> dist{ 0, 100 };
	auto gen = [&dist, &mersenne_engine]() { return dist(mersenne_engine) < 50 ? 0.f : 1.f; };
	generate(h1.begin(), h1.end(), gen);

	float* host_array = h1.data();

	float* device_output;
	hipMalloc(&device_output, w * h * sizeof(float));

	std::ofstream  handler("data/cw1.csv");

	//auto texObj = get_texobject(host_array);

	write_out_result(host_array, handler);
	for (int i = 0; i < number_of_steps; ++i)
	{
		
		auto texObj = get_texobject(host_array);
		
		step(host_array,device_output);
		write_out_result(host_array, handler);
	}

	
	//free(host_array);

	auto err = hipFree(device_output);
	if (err != hipSuccess) { std::cout << "Error freeing array allocation: " << hipGetErrorString(err) << "\n"; return -1; }


}