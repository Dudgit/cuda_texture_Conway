#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "step_funcs.h"
#include "hip/hip_texture_types.h"
#include ""
#include "texture_types.h"

texture<float, 1, hipReadModeElementType> texInput;


__global__ void texture_c(float* output, hipTextureObject_t texobj)
{
	/*
	Inputnak pr�b�lkoztam int1 t�pussal is valamint int array-el, ugyan az a v�geredm�ny
	*/
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; // v�zszintes sorok
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; // f�gg�leges sorok

	output[y * h + x] = tex2D<float>(texobj, x, y-1);
	
}

int main()
{
	// Creating the Vectors
	std::vector<bool> bools(w * h);
	initializer(bools);

	std::vector<float> int_vector(w * h);
	//std::vector<float> output(h * w);
	initializer(int_vector);

	float* hInput = (float*)malloc(sizeof(float) *	h*w);
	float* hOutput = (float*)malloc(sizeof(float) * h*w);
	for (int i = 0; i < h*w; i++)
	{
		hInput[i] = int_vector[i];
	}


	float* dInput = NULL, * dOutput = NULL;

	size_t offset = 0;



	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 0, 0, 0,
			hipChannelFormatKindFloat);
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, w, h);

	auto err = hipMemcpyToArray(cuArray, 0, 0, hInput, w * h * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;


	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);


	float* output;
	hipMalloc(&output, w * h * sizeof(float));

	dim3 dimBlock(w / block_size, h / block_size);
	dim3 dimGrid(block_size, block_size);
	texture_c <<< dimGrid,dimBlock >> > (output, texObj);

	
	//std::cout << "Succes rages on" << std::endl;
	err = hipMemcpy(hOutput, output, w * h * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }
	
	for (int x = 0; x < h; ++x)
	{
		for (int y = 0; y < w; ++y)
		{
			std::cout << int_vector[x * h + y]<<' ';
		}
		std::cout << std::endl;
	}
	
	for (int i = 0; i < h * w; ++i)
	{
		std::cout << hOutput[i] <<  std::endl;
	}
}