#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "step_funcs.h"
#include "hip/hip_texture_types.h"
#include ""
#include "texture_types.h"

texture<float, 1, hipReadModeElementType> texInput;


__global__ void texture_c(float* output)
{
	/*
	Inputnak pr�b�lkoztam int1 t�pussal is valamint int array-el, ugyan az a v�geredm�ny
	*/
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	//output[y * h + x] = tex2D<float>(input, x, y);
	output[y * h + x] = tex1Dfetch<float>(texInput, y * h + x);
	
	
	
}

int main()
{
	// Creating the Vectors
	std::vector<bool> bools(w * h);
	initializer(bools);

	std::vector<float> int_vector(w * h);
	std::vector<float> output(h * w);
	initializer(int_vector);

	float* hInput = (float*)malloc(sizeof(float) *	h*w);
	float* hOutput = (float*)malloc(sizeof(float) * h*w);
	for (int i = 0; i < h*w; i++)
	{
		hInput[i] = int_vector[i];
	}


	float* dInput = NULL, * dOutput = NULL;

	size_t offset = 0;

	texInput.addressMode[0] = hipAddressModeBorder;
	texInput.addressMode[1] = hipAddressModeBorder;
	texInput.filterMode = hipFilterModePoint;
	texInput.normalized = false;


	hipError_t err = (hipMalloc((void**)&dInput, sizeof(float) *h*w));
	if (err != hipSuccess) { std::cout << "Error allocating Cuda memory: " << hipGetErrorString(err) << '\n'; return -1; }

	err = (hipMalloc((void**)&dOutput, sizeof(float) * h * w));
	if (err != hipSuccess) { std::cout << "Error allocating Cuda memory: " << hipGetErrorString(err) << '\n'; return -1; }

	hipMemcpy(dInput, hInput, sizeof(float) * h*w, hipMemcpyHostToDevice);

	hipBindTexture(&offset, texInput, dInput, sizeof(float) * h*w);



	dim3 dimGrid(w / block_size, h / block_size);
	dim3 dimBlock(block_size, block_size);

	texture_c <<< dimGrid, dimBlock >>> (dOutput);

	//std::cout << "Succes rages on" << std::endl;
	err = hipMemcpy(hOutput, dOutput, w * h * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }
	for (int i = 0; i < h * w; ++i)
	{
		std::cout << hOutput[i] << " " << int_vector[i] << std::endl;
	}
}