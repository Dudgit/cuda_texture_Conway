#include "cuda_utility.cuh"
#include <windows.h>
int main()
{
	

	int number_of_steps = 30;
	//Initialize data holder vectors

	std::vector<bool> reff_vec(h * w);
	std::vector<int> gpu_vec(h * w);

	//Fill values of the 2 vector
	//generate(reff_vec.begin(), reff_vec.end(), rg::gen);
	//testing with glidder
	reff_vec =
	{ 
		0,0,0,0,0,0,
		0,0,0,0,1,0,
		0,0,1,0,1,0,
		0,0,0,1,1,0,
		0,0,0,0,0,0,
		0,0,0,0,0,0

	}; 
	std::copy(reff_vec.begin(), reff_vec.end(), gpu_vec.begin());

	//Creating the table object
	table t1(h, w, reff_vec);

	//Creating arrays for host and device
	int* host_array = gpu_vec.data();
	int* device_output;
	hipMalloc(&device_output, w * h * sizeof(int));

	// Creating the texture object
	hipArray* cuArray;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
	error_check(hipMallocArray(&cuArray, &channelDesc, w, h) ,"allocating memory");
	error_check(hipMemcpyToArray(cuArray, 0, 0, host_array, w * h * sizeof(int), hipMemcpyHostToDevice), "copying memor to device");

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;


	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	
	//Output data
	std::ofstream handler_naive("data/naive_conway.txt");
	std::ofstream handler_gpu("data/texture_conway.txt");

	//Write out, the initial table
	write_out_result(host_array, handler_gpu);
	t1.write_table_out(handler_naive);

	//Executing the simulation
	for (int i = 0; i < number_of_steps; ++i)
	{
		// One step
		step(host_array,device_output,texObj,cuArray);
		t1.do_game();

		//Write out results
		write_out_result(host_array, handler_gpu);
		t1.write_table_out(handler_naive);
	}


	//Free the allocated memory
	error_check(hipFree(device_output), "freeing array");

	//Destroy cuda object
	error_check(hipDestroyTextureObject(texObj), "destroying cuda texture");
	
	error_check(hipFreeArray(cuArray), "freeing Cuda array");
	handler_gpu.close();
	handler_naive.close();
	
	//Saveing configurations
	std::ofstream cfg("cfg.txt");
	cfg << number_of_steps << ' ' << w << ' ' << h;
	cfg.close();
}