#include "cuda_utility.cuh"




int main()
{

	int number_of_steps =1000;// (int)argv[1];
	std::vector<float> h1(h * w);
	/*	{
		0,0,0,0,0,0,
		0,0,1,0,0,0,
		1,0,1,0,0,0,
		0,1,1,0,0,0,
		0,0,0,0,0,0,
		0,0,0,0,0,0

	};*/
	

	std::random_device rd{};
	std::mt19937 mersenne_engine{ rd() };  // Generates random integers
	std::uniform_real_distribution<float> dist{ 0, 100 };
	auto gen = [&dist, &mersenne_engine]() { return dist(mersenne_engine) < 50 ? 0.f : 1.f; };
	generate(h1.begin(), h1.end(), gen);

	float* host_array = h1.data();
	std::ofstream  handler("data/cw1.csv");

	auto texObj = get_texobject(host_array);
	
	float* device_output;
	hipMalloc(&device_output, w * h * sizeof(float));



	write_out_result(host_array,handler);
	for (int i = 0; i < number_of_steps; ++i)
	{
		run_kernel(device_output, texObj, host_array, h, w);
		write_out_result(host_array, handler);
	}



	auto err = hipDestroyTextureObject(texObj);
	if (err != hipSuccess) { std::cout << "Error destroying texture object: " << hipGetErrorString(err) << "\n"; return -1; }

	free(host_array);
	free(device_output);
}