#include "cuda_utility.cuh"




int main(int argc,char* argv[])
{

	int number_of_steps = 3;// (int)argv[1];
	float* host_array = new float(h * w);
	initializer(host_array);
	std::ofstream  handler("data/Conway.txt");
	write_out_result(host_array,handler);
	for (int i = 0; i < number_of_steps; ++i)
	{
		step(host_array);
		write_out_result(host_array, handler);
	}
}