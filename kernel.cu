#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "step_funcs.h"
#include "hip/hip_texture_types.h"
#include ""
#include "texture_types.h"




__global__ void texture_c(int* output, hipTextureObject_t input)
{
	/*
	Inputnak pr�b�lkoztam int1 t�pussal is valamint int array-el, ugyan az a v�geredm�ny
	*/
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	output[y * h + x] = tex2D<int>(input, x, y);
}

int main()
{
	// Creating the Vectors
	std::vector<bool> bools(w*h); 
	initializer(bools);

	std::vector<int> int_vector(w*h);
	std::vector<int> output(h*w);
	initializer(int_vector);


	//Hab�r itt integereket haszn�lok, ha �t�rom a Floatot nem m�k�dik
	hipChannelFormatDesc channelDescInput = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* aInput;

	hipError_t err = hipMallocArray(&aInput, &channelDescInput, w, h);
	if (err != hipSuccess) { std::cout << "Error allocating Cuda memory: " << hipGetErrorString(err) << '\n'; return -1; }
	
	
	err = hipMemcpyToArray(aInput, 0, 0, int_vector.data(), w * h * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) { std::cout << "Error copying memory to device: " << hipGetErrorString(err) << '\n'; return -1; }
	
	hipResourceDesc resdescInput{};
	resdescInput.resType = hipResourceTypeArray;
	resdescInput.res.array.array = aInput;

	hipTextureDesc texD{};
	texD.addressMode[0] = hipAddressModeBorder;
	texD.addressMode[1] = hipAddressModeBorder;
	texD.filterMode = hipFilterModeLinear;
	texD.readMode = hipReadModeElementType;
	texD.normalizedCoords = 0;


	hipTextureObject_t t_handler = 0;
	err = hipCreateTextureObject(&t_handler, &resdescInput, &texD, nullptr);
	if (err != hipSuccess) { std::cout << "Error creating texture object : " << hipGetErrorString(err) << '\n'; return -1; }


	int* c_output;
	//int* c_output = new int[h*w];
	err = hipMalloc((void**)&c_output, w * h * sizeof(int));
	if (err != hipSuccess) { std::cout << "Error allocating CUDA memory: " << hipGetErrorString(err) << '\n'; return -1; }
	


	dim3 dimGrid(w / block_size, h / block_size);
	dim3 dimBlock(block_size, block_size);

	texture_c <<< dimGrid,  dimBlock>>> (c_output, t_handler);

	//std::cout << "Succes rages on" << std::endl;
	err = hipMemcpy(output.data(),c_output, w * h * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) { std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }
	for (auto e:output)
	{
		  std::cout << e << std::endl; 
	}

	/*
	table f_table(w,h,bs);
	f_table.write_table_out();
	std::cout << std::endl;
	step(f_table);
	f_table.write_table_out();
	*/
}	
